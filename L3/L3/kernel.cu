#include "hip/hip_runtime.h"
﻿#include "hip/hip_runtime.h"
#include ""

#include <stdio.h>
#include <stdlib.h>
#include <string.h>
#include <iostream>
#include <fstream>
#include <vector>
#include "nlohmann/json.hpp"

using json = nlohmann::json;
using namespace std;

const int RESULT_SIZE = 30;

struct Horse {
	char Name[256];
	int Age;
	float Speed;
};

__device__ char toUpperCase(char c) {
	if (c >= 'a' && c <= 'z') {
		return c - ('a' - 'A');
	}
	return c;
}

__device__ const char* getAgeCategory(int age) {
	if (age > 10) {
		return "Senas";
	}
	else {
		return "Jaunas";
	}
}

__device__ void myStrCat(char* dest, const char* src, int destSize) {
	int destLen = 0;
	while (dest[destLen] != '\0' && destLen < destSize) {
		++destLen;
	}

	if (destLen > 0 && destLen < destSize - 1) {
		dest[destLen] = '-';
		++destLen;
	}

	int srcLen = 0;
	if (destLen < destSize - 1 && destLen > 0) {
		dest[destLen] = ' ';
		++destLen;
	}

	while (src[srcLen] != '\0') {
		if (destLen < destSize - 1) {
			dest[destLen] = src[srcLen];
			++destLen;
		}
		++srcLen;
	}
	dest[destLen] = '\0';
}

__global__ void processDataKernel(Horse* horses, char* results, int dataSize, int* index) {
	int threadIndex = threadIdx.x + blockIdx.x * blockDim.x;
	char temp_result[RESULT_SIZE];

	if (threadIndex < dataSize) {
		Horse currentHorse = horses[threadIndex];

		for (int i = 0; i < RESULT_SIZE; ++i) {
			if (currentHorse.Name[i] != '\0') {
				temp_result[i] = toUpperCase(currentHorse.Name[i]);
			}
			else {
				temp_result[i] = ' ';
				break;
			}
		}

		const char* AgeCategory = getAgeCategory(currentHorse.Age);
		myStrCat(temp_result, AgeCategory, RESULT_SIZE);

		if (temp_result[0] == 'X') {
			int writeIndex = atomicAdd(index, RESULT_SIZE);
			for (int i = 0; i < RESULT_SIZE; ++i) {
				results[writeIndex + i] = temp_result[i];
			}
		}
	}
}

int main() {
	// Read data from the file
	std::ifstream inputFile("IFF-1-8_BriaukaE_L1_dat_1.json"); //All data that follows criteria
	
	if (!inputFile.is_open()) {
		std::cerr << "Error opening the input file." << std::endl;
		return 1;
	}

	json jsonData;
	inputFile >> jsonData;
	inputFile.close();

	if (!jsonData.is_array() || jsonData.empty()) {
		std::cerr << "Invalid or empty JSON file." << std::endl;
		return 1;
	}

	std::vector<Horse> horses;
	for (const auto& horseData : jsonData) {
		Horse horse;

		strncpy(horse.Name, horseData["Name"].get<std::string>().c_str(), sizeof(horse.Name) - 1);
		horse.Name[sizeof(horse.Name) - 1] = '\0';

		horse.Age = horseData["Age"];
		horse.Speed = horseData["Speed"];
		horses.push_back(horse);
	}


	int dataSize = horses.size();
	if (dataSize == 0) {
		std::cerr << "No data found in the input file." << std::endl;
		return 1;
	}

	// Allocate memory for horses on the device
	Horse* d_horses;
	hipMalloc((void**)&d_horses, dataSize * sizeof(Horse));

	// Copy input data from host to device
	hipMemcpy(d_horses, horses.data(), dataSize * sizeof(Horse), hipMemcpyHostToDevice);

	// Allocate memory for results on the device
	char* d_results;
	hipMalloc((void**)&d_results, dataSize * RESULT_SIZE * sizeof(char));
	hipMemset(d_results, ' ', dataSize * RESULT_SIZE * sizeof(char));

	// Allocate memory for index on the device
	int* d_index;
	hipMalloc((void**)&d_index, sizeof(int));
	hipMemset(d_index, 0, sizeof(int));

	// Launch the CUDA kernel
	int threadsperBlock = 32;
	int numBlocks = (dataSize + threadsperBlock - 1) / threadsperBlock;
	processDataKernel<<<numBlocks, threadsperBlock>>>(d_horses, d_results, dataSize, d_index);

	// Copy index from device to host
	int index;
	hipMemcpy(&index, d_index, sizeof(int), hipMemcpyDeviceToHost);

	// Copy results from device to host
	char* host_results = new char[index];
	hipMemcpy(host_results, d_results, index * sizeof(char), hipMemcpyDeviceToHost);
	hipDeviceSynchronize();

	//Writing char result to file
	std::ofstream outputFile("IFF-1-8_ErikasB_L1_rez_1.txt");
	if (!outputFile.is_open()) {
		std::cerr << "Error opening the output file." << std::endl;
		return 1;
	}
	for (int i = 0; i < index; i++) {
		char a = host_results[i];
		if (a != '\0') {
			outputFile << a;
		}
		if ((i + 1) % RESULT_SIZE == 0 && i != index - 1) {
			outputFile << "\n";
		}
	}
	outputFile.close();

	// Free allocated memory
	delete[] host_results;
	hipFree(d_horses);
	hipFree(d_results);
	hipFree(d_index);

	return 0;
}